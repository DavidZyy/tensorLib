#include "Tensor.hpp"
#include "device/cuda/CUDA.cuh"
#include "test.hpp"

#define M 1
// #define N (16)
// #define K (16)
// #define N (4096 * 1)
// #define K (4096 * 1)
#define N (11008)
#define K (11008)

/************************************************************************************************************************************************************/

template<typename dtype> void gemv_v0(const dtype* A, const dtype* B, dtype* C, size_t n, size_t k);
template<typename dtype> void gemv_v3(const dtype* A, const dtype* B, dtype* C, size_t n, size_t k);
template<typename dtype> void gemv_v4(const dtype* A, const dtype* B, dtype* C, size_t n, size_t k);
template<typename dtype> void gemv_v5(const dtype* A, const dtype* B, dtype* C, size_t n, size_t k);
template<typename dtype> void gemv_v6(const dtype* A, const dtype* B, dtype* C, size_t n, size_t k);
template<typename dtype> void gemv_v7(const dtype* A, const dtype* B, dtype* C, size_t n, size_t k);
template<typename dtype> void gemv_v8(const dtype* A, const dtype* B, dtype* C, size_t n, size_t k);
// template<typename dtype> void gemv_v9(const dtype* A, const dtype* B, dtype* C, size_t n, size_t k);
template<typename dtype> void gemv_cublasSgemv(const dtype* A, const dtype* B, dtype* C, size_t n, size_t k);

/************************************************************************************************************************************************************/

template<typename dtype, typename GemvFunc>
float timer(GemvFunc gemv, Tensor<dtype> A, Tensor<dtype> B, Tensor<dtype> C, int repeat) {
    hipEvent_t start, end;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&end));
    CUDA_CHECK(hipEventRecord(start));
    for (int i = 0; i < repeat; i++)
        gemv(A.device->getDataPtr(), B.device->getDataPtr(), C.device->getDataPtr(), N, K);
    CUDA_CHECK(hipEventRecord(end));
    CUDA_CHECK(hipEventSynchronize(end));

    float msec, sec;
    CUDA_CHECK(hipEventElapsedTime(&msec, start, end));
    sec = msec / 1000.0;
    return sec;
}

template<typename dtype, typename GemvFunc>
void profile(GemvFunc gemv, Tensor<dtype> A, Tensor<dtype> B, Tensor<dtype> C, Tensor<dtype> C_r, int repeat, std::string name) {
    std::cout << "------------------" << name << "------------------" << std::endl;

    float total_sec = timer(gemv, A, B, C, repeat);

    double avg_sec = total_sec / repeat;
    double avg_Gflops = ((double)M) * N * K * 2 / 1024 / 1024 / 1024 / avg_sec;

    std::cout << "average time: " << avg_sec << "s" << std::endl;
    std::cout << "average Gflops: " << avg_Gflops << std::endl;

    if (check_equal(C, C_r)) {
        std::cout << "pass!" << std::endl;
    } else {
        std::cout << "failed!" << std::endl;
    }
}

/************************************************************************************************************************************************************/

int main() {
    Tensor<half> A = randn<half>({M, K}, "cuda");
    Tensor<half> B = randn<half>({K, N}, "cuda");
    // Tensor<half> A = full<half>({M, K}, 1, "cuda");
    // Tensor<half> B = full<half>({K, N}, 1, "cuda");
    Tensor<half> C = A.matmul(B.transpose(0, 1)); // B is col major, a little mess ... We can think that C is correctness

    Tensor<half> C0 = full<half>({M, N}, 0, "cuda");
    Tensor<half> C3 = full<half>({M, N}, 0, "cuda");
    Tensor<half> C4 = full<half>({M, N}, 0, "cuda");
    Tensor<half> C6 = full<half>({M, N}, 0, "cuda");
    Tensor<half> C7 = full<half>({M, N}, 0, "cuda");
    Tensor<half> C8 = full<half>({M, N}, 0, "cuda");
    // Tensor<half> C9 = full<half>({M, N}, 0, "cuda");

    int repeat = 100;

    profile(gemv_v0<half>, A, B, C0, C, repeat, "gemv_v0");
    profile(gemv_v3<half>, A, B, C3, C, repeat, "gemv_v3");
    profile(gemv_v4<half>, A, B, C4, C, repeat, "gemv_v4");
    profile(gemv_v6<half>, A, B, C6, C, repeat, "gemv_v6");
    // profile(gemv_v7<half>, A, B, C7, C, repeat, "gemv_v7");
    profile(gemv_v8<half>, A, B, C8, C, repeat, "gemv_v8");
    // profile(gemv_v9<half>, A, B, C8, C, repeat, "gemv_v9");

}

// int main() {
//     Tensor<float> A = randn<float>({M, K}, "cuda");
//     Tensor<float> B = randn<float>({K, N}, "cuda");
//     // Tensor<float> A = full<half>({M, K}, 1, "cuda");
//     // Tensor<float> B = full<half>({K, N}, 1, "cuda");
//     Tensor<float> C = A.matmul(B.transpose(0, 1)); // B is col major, a little mess ... We can think that C is correctness
// 
//     Tensor<float> C0 = full<float>({M, N}, 0, "cuda");
//     Tensor<float> C3 = full<float>({M, N}, 0, "cuda");
//     Tensor<float> C4 = full<float>({M, N}, 0, "cuda");
//     Tensor<float> C5 = full<float>({M, N}, 0, "cuda");
//     Tensor<float> C_cublasSgemv = full<float>({M, N}, 0, "cuda");
// 
//     int repeat = 50;
// 
//     profile(gemv_v0<float>, A, B, C0, C, repeat, "gemv_v0");
//     profile(gemv_v3<float>, A, B, C3, C, repeat, "gemv_v3");
//     profile(gemv_v4<float>, A, B, C4, C, repeat, "gemv_v4");
//     profile(gemv_v5<float>, A, B, C5, C, repeat, "gemv_v5");
//     profile(gemv_cublasSgemv<float>, A, B, C_cublasSgemv, C, repeat, "gemv_cublasSgemv");
// }
