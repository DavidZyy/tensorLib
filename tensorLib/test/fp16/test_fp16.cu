#include "hip/hip_runtime.h"

/**
    NOTE: this file mainly check the correctness of fp16 in cuda, in cpu not been checked.
 */
#include "Tensor.hpp"
#include "device/CUDA.hpp"
#include "nn/rmsNorm.hpp"
#include <iostream>
#include <chrono>
#include "Transformer.hpp"

#define MEASURE_TIME(code_block) \
    do { \
        auto start = std::chrono::high_resolution_clock::now(); \
        code_block; \
        auto end = std::chrono::high_resolution_clock::now(); \
        auto duration = std::chrono::duration<double>(end - start); \
        std::cout << "Execution time: " << duration.count() << " seconds" << std::endl; \
    } while (0)

/************************************************************************************************************************************************************/

// Helper CUDA kernel to compare tensors element-wise
template <typename T>
__global__ void compare_kernel(const T* a_data, const T* b_data, size_t num_elements, bool* result_flag) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_elements) {
        // Convert half to float for comparison
        float a = __half2float(a_data[idx]);
        float b = __half2float(b_data[idx]);

        // Compare the elements, and if they are different, set the flag
        if (fabs(a - b) >= 1e-1) {
            printf("idx: %d, a: %f, b: %f\n", idx, a, b);
            *result_flag = false; // Set result flag to false if difference is found
        }
    }
}

// Function to check equality of two tensors
template <typename T>
bool check_equal(const Tensor<T>& a, const Tensor<T>& b) {
    if (a.num_elements != b.num_elements) {
        std::cerr << "Tensors have different sizes!" << std::endl;
        return false;
    }

    // Allocate memory for a flag on the device to indicate equality
    bool* d_result_flag;
    bool h_result_flag = true; // Initialize to true (assume equality)
    hipMalloc(&d_result_flag, sizeof(bool));
    hipMemcpy(d_result_flag, &h_result_flag, sizeof(bool), hipMemcpyHostToDevice);

    // Launch kernel to compare the tensors
    int block_size = 256; // Choose an appropriate block size
    int grid_size = (a.num_elements + block_size - 1) / block_size;
    compare_kernel<<<grid_size, block_size>>>(a.device->getDataPtr(), b.device->getDataPtr(), a.num_elements, d_result_flag);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    // Copy the result flag back to the host
    hipMemcpy(&h_result_flag, d_result_flag, sizeof(bool), hipMemcpyDeviceToHost);

    // Free the device memory for the flag
    hipFree(d_result_flag);

    return h_result_flag; // Return whether the tensors are equal
}

/************************************************************************************************************************************************************/

__device__ static float atomicMax(float* address, float val) {
    int* address_as_int = (int*)address;
    int old = *address_as_int, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_int, assumed, __float_as_int(fmaxf(val, __int_as_float(assumed))));
    } while (assumed != old);
    return __int_as_float(old);
}

template <typename T>
__global__ void compare_max_kernel(const T* a_data, const T* b_data, size_t num_elements, float* max_diff) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int thread_id = threadIdx.x;

    // Allocate shared memory for reduction within a block
    __shared__ float shared_max[256];  // Assuming block size of 256

    // Initialize the shared memory
    if (thread_id < 256) {
        shared_max[thread_id] = 0.0f;
    }
    __syncthreads();

    // Perform computation for each element
    if (idx < num_elements) {
        // Convert half to float for comparison
        float a = __half2float(a_data[idx]);
        float b = __half2float(b_data[idx]);

        // Calculate the absolute difference
        float diff = fabs(a - b);

        // Store the difference in the shared memory
        shared_max[thread_id] = diff;
    }
    __syncthreads();

    // Perform block-wide reduction to find the max difference within this block
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (thread_id < s) {
            shared_max[thread_id] = fmaxf(shared_max[thread_id], shared_max[thread_id + s]);
        }
        __syncthreads();
    }

    // The thread with thread_id == 0 will write the result to global memory
    if (thread_id == 0) {
        atomicMax(max_diff, shared_max[0]);
    }
}

template <typename T>
float get_max_abs_difference(const Tensor<T>& a, const Tensor<T>& b) {
    if (a.num_elements != b.num_elements) {
        std::cerr << "Tensors have different sizes!" << std::endl;
        return -1.0f; // Return an error value if tensors are not equal in size
    }

    // Allocate memory for max_diff on the device
    float* d_max_diff;
    float h_max_diff = 0.0f; // Initialize to 0 (no difference yet)
    hipMalloc(&d_max_diff, sizeof(float));
    hipMemcpy(d_max_diff, &h_max_diff, sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel to compare the tensors and track the max absolute difference
    int block_size = 256; // Choose an appropriate block size (must be a power of 2)
    int grid_size = (a.num_elements + block_size - 1) / block_size;
    compare_max_kernel<<<grid_size, block_size>>>(a.device->getDataPtr(), b.device->getDataPtr(), a.num_elements, d_max_diff);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    // Copy the max difference back to the host
    hipMemcpy(&h_max_diff, d_max_diff, sizeof(float), hipMemcpyDeviceToHost);

    // Free the device memory for max_diff
    hipFree(d_max_diff);

    return h_max_diff; // Return the maximum absolute difference
}

/************************************************************************************************************************************************************/

template <typename T>
void check_equal_and_max_diff(Tensor<T>& a, Tensor<T>& b) {
    if (check_equal(a, b)) {
        std::cout << "pass!" << std::endl;
    } else {
        std::cout << "failed!" << std::endl;
    }
    float max_diff = get_max_abs_difference(a, b);
    std::cout << "max_diff: " << max_diff << std::endl;
}

/************************************************************************************************************************************************************/

void test_matmul() {
    int bs = 1;
    // int m = 128;
    int m = 1;
    int n = 4096;
    int k = 4096;

    Tensor<float> a = randn<float>({bs, m, k}, "cuda");
    Tensor<float> b = randn<float>({bs, k, n}, "cuda");
    Tensor<float> c;
    MEASURE_TIME({
        c = a.matmul(b); 
    });

    // std::cout << "a: " << std::endl << a << std::endl;
    // std::cout << "b: " << std::endl << b << std::endl;
    // std::cout << "c: " << std::endl << c << std::endl;

    Tensor<half> a_fp16(a);
    Tensor<half> b_fp16(b);
    Tensor<half> _c_fp16;
    MEASURE_TIME({
        _c_fp16 = a_fp16.matmul(b_fp16);
    });

    // std::cout << "c: " << std::endl << c << std::endl;
    // std::cout << "c_fp16: " << std::endl << c_fp16 << std::endl;
    // std::cout << "_c_fp16: " << std::endl << _c_fp16 << std::endl;

    Tensor<half> c_fp16(c);
    check_equal_and_max_diff(c_fp16, _c_fp16);
}

/************************************************************************************************************************************************************/

void test_rms() {
    int n = 1;
    int dim = 4096;
    // int dim = 10;
    float eps = 1e-5;

    // fp32
    nn::RMSNorm<float> rms_fp32 = nn::RMSNorm<float>(dim, eps, "cuda");
    Tensor<float> x = randn<float>({n, dim}, "cuda");
    Tensor<float> y;
    // Tensor<float> y1;
    MEASURE_TIME({
        y = rms_fp32.forward_fused_cuda(x);
        // y = rms_fp32.forward_plain(x);
        // y1 = rms_fp32.forward_plain(x);
    });

    std::cout << "x: " << std::endl << x << std::endl;
    std::cout << "weight:" << std::endl << rms_fp32.weight << std::endl;
    std::cout << "y: " << std::endl << y << std::endl;
    // std::cout << "y1: " << std::endl << y1 << std::endl;

    // fp16
    nn::RMSNorm<half> rms_fp16 = nn::RMSNorm<half>(dim, eps, "cuda");
    // set weight
    Tensor<half> weight_fp16(rms_fp32.weight);
    rms_fp16.weight = weight_fp16;

    Tensor<half> x_fp16(x);
    Tensor<half> y_fp16;
    MEASURE_TIME({
        y_fp16 = rms_fp16.forward_fused_cuda(x_fp16);
        // y_fp16 = rms_fp16.forward_plain(x_fp16);
    });

    std::cout << "x_fp16: " << std::endl << x_fp16 << std::endl;
    std::cout << "weight_fp16:" << std::endl << rms_fp16.weight << std::endl;
    std::cout << "y_fp16: " << std::endl << y_fp16 << std::endl;

    // check equal and get max_diff
    Tensor<half> _y_fp16(y);

    check_equal_and_max_diff(y_fp16, _y_fp16);
}

/************************************************************************************************************************************************************/

// void test_attention() {
//     ModelArgs args;
//     args.n_heads = 128;
//     args.dim = 4096;
//     args.max_batch_size = 1;
//     args.max_seq_len = 2048;
// 
//     // fp32
//     Attention<float> attention_fp32 = Attention<float>(args, "cuda");
//     Tensor<float> x = randn<float>({1, args.dim}, "cuda");
//     Tensor<float> y;
//     MEASURE_TIME({
//         y = attention_fp32.forward(x);
//     });
// 
//     // fp16
// 
// }

/************************************************************************************************************************************************************/

void test_ffn() {
    int dim = 1024;
    int hidden_dim = 1024;
    // int dim = 10;
    // int hidden_dim = 100;

    // fp32
    FeedForward<float> ffn_fp32 = FeedForward<float>(dim, hidden_dim, "cuda");
    // set weight
    Tensor<float> w1 = randn<float>({hidden_dim, dim}, "cuda");
    Tensor<float> w2 = randn<float>({dim, hidden_dim}, "cuda");
    Tensor<float> w3 = randn<float>({hidden_dim, dim}, "cuda");
    ffn_fp32.w1.weight = w1;
    ffn_fp32.w2.weight = w2;
    ffn_fp32.w3.weight = w3;

    Tensor<float> x = randn<float>({1, dim}, "cuda");
    Tensor<float> y;
    MEASURE_TIME({
        y = ffn_fp32.forward(x);
    });

    std::cout << "x: " << std::endl << x << std::endl;
    std::cout << "y: " << std::endl << y << std::endl;

    // fp16
    FeedForward<half> ffn_fp16 = FeedForward<half>(dim, hidden_dim, "cuda");
    // set weight
    Tensor<half> w1_fp16(w1);
    Tensor<half> w2_fp16(w2);
    Tensor<half> w3_fp16(w3);
    ffn_fp16.w1.weight = w1_fp16;
    ffn_fp16.w2.weight = w2_fp16;
    ffn_fp16.w3.weight = w3_fp16;

    Tensor<half> x_fp16(x);
    Tensor<half> y_fp16;
    MEASURE_TIME({
        y_fp16 = ffn_fp16.forward(x_fp16);
    });

    std::cout << "x_fp16: " << std::endl << x_fp16 << std::endl;
    std::cout << "y_fp16: " << std::endl << y_fp16 << std::endl;

    Tensor<half> _y_fp16(y);

    check_equal_and_max_diff(y_fp16, _y_fp16);
}

/************************************************************************************************************************************************************/

void test_argmax() {
    int m = 10;
    int n = 32000;

    Tensor<float> x = randn<float>({m, n}, "cuda");
    Tensor<int> y;
    MEASURE_TIME({
        y = x.argmax(1);
    });

    // std::cout << "x: " << std::endl << x << std::endl;
    // std::cout << "y: " << std::endl << y << std::endl;

    Tensor<half> x_fp16(x);
    Tensor<int> y_fp16;
    MEASURE_TIME({
        y_fp16 = x_fp16.argmax(1);
    });

    // std::cout << "x_fp16: " << std::endl << x_fp16 << std::endl;
    // std::cout << "y_fp16: " << std::endl << y_fp16 << std::endl;

    std::cout << "y: " << std::endl << y << std::endl;
    std::cout << "y_fp16: " << std::endl << y_fp16 << std::endl;

    // check_equal_and_max_diff(y_fp16, y);
}

/************************************************************************************************************************************************************/

int main() {
    // test_matmul();
    // test_rms();
    // test_ffn();
    test_argmax();
    return 0;
}
