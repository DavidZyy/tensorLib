#include "hip/hip_runtime.h"
#include "CUDA.hpp"
#include "Tensor.hpp"
#include <cstddef>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/library_types.h>
#include <stdexcept>
#include <vector>

template class CUDA<float>;
template class CUDA<int>;

template <typename dtype>
CUDA<dtype>::CUDA(size_t size) : Device<dtype>(size) {
    CUDA_CHECK(hipMalloc(&this->data_, size * sizeof(dtype)));
}

template <typename dtype>
CUDA<dtype>::~CUDA() {
    CUDA_CHECK(hipFree(this->data_));
}

/**
 * I think this implementation is not efficient, but it should be good enough for now.
 * We can improve it later.
 * Maybe we can make lhs and rhs be contiguous, and use shared memory in one block ?
 * execuate contiguous in cpu is not efficient, so i give up this step when perform batched matmul in cpu,
 * but in cuda, maybe we can do it ?
 *
 * @tparam dtype 
 */
template <typename dtype>
__global__ void matmulKernel(const dtype* lhs, const dtype* rhs, dtype* result, 
                             const int* lhs_stride, const int* rhs_stride, 
                             size_t lhs_offset, size_t rhs_offset,
                             const int* result_shape, size_t result_elements,
                             size_t K,
                             size_t ndim) {
    // Global thread index for each result element
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= result_elements) return;

    size_t linear_index = idx;
    size_t Aoff = lhs_offset, Boff = rhs_offset;

    // Compute offsets for lhs and rhs
    for (int i = ndim - 1; i >= 0; --i) {
        int cur_dim_id = linear_index % result_shape[i];
        linear_index /= result_shape[i];

        if (i != ndim - 1)
            Aoff += cur_dim_id * lhs_stride[i];
        if (i != ndim - 2)
            Boff += cur_dim_id * rhs_stride[i];
    }

    // Compute the dot product
    dtype sum = 0;
    int t1 = lhs_stride[ndim - 1], t2 = rhs_stride[ndim - 2];
    for (int k = 0; k < K; ++k) {
        sum += lhs[Aoff + k * t1] * rhs[Boff + k * t2];
    }

    // Store the result
    result[idx] = sum;
}

// Wrapper function to launch the CUDA kernel
template <typename dtype>
void CUDA<dtype>::matmul(const dtype* lhs, const dtype* rhs, dtype* result, 
    const std::vector<int>& lhs_stride, 
    const std::vector<int>& rhs_stride, 
    size_t lhs_offset,
    size_t rhs_offset,
    const std::vector<int>& result_shape,
    size_t result_elements,
    size_t K)
{
    int ndim = result_shape.size();

    // Allocate device memory
    int* d_lhs_stride;
    int* d_rhs_stride;
    int* d_result_shape;
    CUDA_CHECK(hipMalloc(&d_lhs_stride, ndim * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_rhs_stride, ndim * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_result_shape, ndim * sizeof(int)));

    // Copy strides and shapes to device memory
    CUDA_CHECK(hipMemcpy(d_lhs_stride, lhs_stride.data(), ndim * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_rhs_stride, rhs_stride.data(), ndim * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_result_shape, result_shape.data(), ndim * sizeof(int), hipMemcpyHostToDevice));

    // Launch the kernel
    int threads_per_block = 256;
    int blocks = (result_elements + threads_per_block - 1) / threads_per_block;
    matmulKernel<<<blocks, threads_per_block>>>(lhs, rhs, result, 
                                                d_lhs_stride, d_rhs_stride, 
                                                lhs_offset, rhs_offset, 
                                                d_result_shape, result_elements, K, ndim);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipFree(d_lhs_stride));
    CUDA_CHECK(hipFree(d_rhs_stride));
    CUDA_CHECK(hipFree(d_result_shape));
}

template <typename dtype>
__global__ void fullKernel(dtype* data, size_t num_elements, dtype fill_value) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_elements) {
        data[idx] = static_cast<dtype>(fill_value);
    }
}

template <typename dtype>
void CUDA<dtype>::full(size_t num_elements, dtype fill_value) {
    int threads_per_block = 256;
    int blocks_per_grid = (num_elements + threads_per_block - 1) / threads_per_block;
    fullKernel<<<blocks_per_grid, threads_per_block>>>(this->data_, num_elements, fill_value);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
}

template <typename dtype>
dtype CUDA<dtype>:: getDataLinear(size_t linear_index) const {
    dtype result;
    CUDA_CHECK(hipMemcpy(&result, this->data_ + linear_index, sizeof(dtype), hipMemcpyDeviceToHost));
    return result;
}

template <typename dtype>
__global__ void contiguous_kernel(
    dtype* result,
    const dtype* data,
    const int* shape,
    const int* stride,
    size_t offset,
    size_t num_elements,
    int dim_size) 
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < num_elements) {
        size_t linear_index_new = 0;
        size_t linear_index = i;

        for (int j = dim_size - 1; j >= 0; --j) {
            int cur_dim_id = linear_index % shape[j];
            linear_index /= shape[j];
            linear_index_new += cur_dim_id * stride[j];
        }

        result[i] = data[linear_index_new + offset];
    }
}

template <typename dtype>
void CUDA<dtype>::contiguous(
    dtype* result,
    const std::vector<int>& shape,
    const std::vector<int>& stride,
    size_t offset,
    size_t num_elements) 
{
    // Allocate memory for shape and stride on the device
    int* d_shape;
    int* d_stride;
    hipMalloc(&d_shape, shape.size() * sizeof(int));
    hipMalloc(&d_stride, stride.size() * sizeof(int));

    // Copy shape and stride data to device
    hipMemcpy(d_shape, shape.data(), shape.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_stride, stride.data(), stride.size() * sizeof(int), hipMemcpyHostToDevice);

    // Calculate grid and block dimensions
    int threads_per_block = 256;
    int num_blocks = (num_elements + threads_per_block - 1) / threads_per_block;

    // Launch the kernel
    contiguous_kernel<<<num_blocks, threads_per_block>>>(
        result, this->data_, d_shape, d_stride, offset, num_elements, shape.size());

    // Free device memory for shape and stride
    hipFree(d_shape);
    hipFree(d_stride);
}
