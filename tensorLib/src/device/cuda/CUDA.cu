#include "hip/hip_runtime.h"
#include "device/CUDA.hpp"
#include <cstddef>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/library_types.h>
#include <iostream>
#include <vector>
#include <hiprand/hiprand_kernel.h>

template class CUDA<int8_t>;
template class CUDA<half>;
template class CUDA<float>;
template class CUDA<int>;

template <typename dtype>
CUDA<dtype>::CUDA(size_t size) : Device<dtype>(size) {
    CUDA_CHECK(hipMalloc(&this->data_, size * sizeof(dtype)));
}

// template <typename dtype>
// CUDA<dtype>::CUDA(size_t size) : Device<dtype>(size) {
//     // Allocate memory aligned to 16 bytes
//     CUDA_CHECK(cudaMallocAligned(&this->data_, size * sizeof(dtype), 16));
// }

template <typename dtype>
CUDA<dtype>::~CUDA() {
    // free a null prt get error: code: 4, reason: driver shutting down
    if(this->data_ != nullptr)
        CUDA_CHECK(hipFree(this->data_));
}

template <typename dtype>
__global__ void fullKernel(dtype* data, size_t num_elements, dtype fill_value) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_elements) {
        data[idx] = static_cast<dtype>(fill_value);
    }
}

template <typename dtype>
void CUDA<dtype>::full(size_t num_elements, dtype fill_value) {
    int threads_per_block = 256;
    int blocks_per_grid = (num_elements + threads_per_block - 1) / threads_per_block;
    fullKernel<<<blocks_per_grid, threads_per_block>>>(this->data_, num_elements, fill_value);
    CUDA_CHECK(hipGetLastError());
    // CUDA_CHECK(hipDeviceSynchronize());
}

template <typename dtype>
__global__ void randnKernel(dtype* data, size_t num_elements, unsigned long long seed) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < num_elements) {
        hiprandState state;
        unsigned long long thread_seed = seed + idx + clock64(); // Timestamp + idx to ensure unique seeds
        hiprand_init(thread_seed + idx, idx, 0, &state);
        data[idx] = hiprand_normal(&state);
    }
}

template <typename dtype>
void CUDA<dtype>::randn(size_t num_elements) {
    int threads_per_block = 256;
    int blocks_per_grid = (num_elements + threads_per_block - 1) / threads_per_block;
    unsigned long long seed = time(NULL); // Get a time-based seed
    randnKernel<<<blocks_per_grid, threads_per_block>>>(this->data_, num_elements, seed);
}

template <typename dtype>
dtype CUDA<dtype>::getDataLinear(size_t linear_index) const {
    dtype result;
    CUDA_CHECK(hipMemcpy(&result, this->data_ + linear_index, sizeof(dtype), hipMemcpyDeviceToHost));
    return result;
}

template <typename dtype>
void CUDA<dtype>::setDataLinear(size_t linear_index, dtype value) {
    CUDA_CHECK(hipMemcpy(this->data_ + linear_index, &value, sizeof(dtype), hipMemcpyHostToDevice));
}

template <typename dtype>
__global__ void contiguous_kernel(
    dtype* result,
    const dtype* data,
    CudaVec shape,
    CudaVec stride,
    size_t offset,
    size_t num_elements)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < num_elements) {
        size_t linear_index_new = convertIdx(i, shape, stride, offset);
 
        result[i] = data[linear_index_new];
    }
}

template <typename dtype>
void CUDA<dtype>::contiguous(
    dtype* result,
    const std::vector<int>& shape,
    const std::vector<int>& stride,
    size_t offset,
    size_t num_elements) 
{
    // Calculate grid and block dimensions
    int threads_per_block = 256;
    int num_blocks = (num_elements + threads_per_block - 1) / threads_per_block;

    // Launch the kernel
    contiguous_kernel<<<num_blocks, threads_per_block>>>(
        result, this->data_, VecToCuda(shape), VecToCuda(stride), offset, num_elements);
    CUDA_CHECK(hipGetLastError());
}

template <typename dtype>
__global__ void setItemEwiseKernel(
    dtype* data,
    const dtype* src,
    CudaVec shape,
    CudaVec stride, 
    size_t offset,
    size_t num_elements) 
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < num_elements) {
        size_t linearIdx = convertIdx(i, shape, stride, offset);
        data[linearIdx] = src[i];
    }
}

template <typename dtype>
void CUDA<dtype>::setItemEwise(
    dtype* src,
    const std::vector<int>& shape,
    const std::vector<int>& stride,
    size_t offset,
    size_t num_elements) 
{
    // Define grid and block dimensions
    int blockSize = 256;
    int gridSize = (num_elements + blockSize - 1) / blockSize;

    // Launch the kernel
    setItemEwiseKernel<<<gridSize, blockSize>>>(
        this->data_, src, VecToCuda(shape), VecToCuda(stride), offset, num_elements);

    CUDA_CHECK(hipGetLastError());
}

template <typename dtype>
__global__ void setItemScalarKernel(
    dtype* data,
    const dtype value,
    CudaVec shape,
    CudaVec stride, 
    size_t offset,
    size_t num_elements) 
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < num_elements) {
        size_t linearIdx = convertIdx(i, shape, stride, offset);
        data[linearIdx] = value;
    }
}

template <typename dtype>
void CUDA<dtype>::setItemScalar(
    dtype value,
    const std::vector<int>& shape,
    const std::vector<int>& stride,
    size_t offset,
    size_t num_elements) 
{
    // Define grid and block dimensions
    int blockSize = 256;
    int gridSize = (num_elements + blockSize - 1) / blockSize;

    // Launch the kernel
    setItemScalarKernel<<<gridSize, blockSize>>>(
        this->data_, value, VecToCuda(shape), VecToCuda(stride), offset, num_elements);

    CUDA_CHECK(hipGetLastError());
}

////////////////////////////////////////////////////// unary operations ///////////////////////////////////////////////////////////////////////////////
/**
 * pass function pointer like below have bug,
 * __global__ void unaryKernel(dtype* result, const dtype* src, size_t num_elements, dtype (*op)(dtype)),
 * seems should use hipMemcpyFromSymbol first, so I use template instead.
 */
template <typename dtype, dtype (*op)(dtype)>
__global__ void unaryKernel(dtype* result, const dtype* src, size_t num_elements) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < num_elements) {
        result[i] = op(src[i]);
    }
}

template <typename dtype>
template <dtype (*op)(dtype)>
void CUDA<dtype>::applyUnaryOperation(dtype* result, size_t num_elements) const {
    int blockSize = 256;  // Number of threads per block (adjust based on optimization needs)
    int gridSize = (num_elements + blockSize - 1) / blockSize;  // Number of blocks
    unaryKernel<dtype, op><<<gridSize, blockSize>>>(result, this->data_, num_elements);
    CUDA_CHECK(hipGetLastError());
    // CUDA_CHECK(hipDeviceSynchronize());
}

template <typename dtype>
__device__ dtype negateFunc(dtype x) {
    return -x;
}

template <typename dtype>
void CUDA<dtype>::neg(dtype* result, size_t num_elements) {
    applyUnaryOperation<negateFunc<dtype>>(result, num_elements);
}

// template <typename dtype>
// __device__ dtype sinFunc(dtype x) {
//     if constexpr (std::is_same<dtype, int>::value || std::is_same<dtype, int8_t>::value) {
//         // If dtype is an integer type, cast x to float and calculate sine, or it will link to std::sin, which is not supported on CUDA
//         return static_cast<dtype>(sin(static_cast<float>(x)));
//     } else {
//         return sin(x);
//     }
// }

template <typename dtype>
__device__ dtype sinFunc(dtype x) {
    if constexpr (std::is_same<dtype, int>::value || std::is_same<dtype, int8_t>::value) {
        // If dtype is an integer type, cast x to float and calculate sine, or it will link to std::sin, which is not supported on CUDA
        return static_cast<dtype>(sin(static_cast<float>(x)));
    } else if constexpr (std::is_same<dtype, __half>::value) {
        // If dtype is __half (half precision), use the CUDA-specific __hsin
        return __float2half(sinf(__half2float(x)));
    } else {
        // For other types (float, double), use standard sin
        return sin(x);
    }
}


template <typename dtype>
void CUDA<dtype>::sin(dtype* result, size_t num_elements) {
    applyUnaryOperation<sinFunc<dtype>>(result, num_elements);
}

template <typename dtype>
__device__ dtype cosFunc(dtype x) {
    if constexpr (std::is_same<dtype, int>::value || std::is_same<dtype, int8_t>::value) {
        return static_cast<dtype>(cos(static_cast<float>(x)));
    } else if constexpr (std::is_same<dtype, __half>::value) {
        // If dtype is __half (half precision), use the CUDA-specific __hsin
        return __float2half(cosf(__half2float(x)));
    } else {
        return cos(x);
    }
}

template <typename dtype>
void CUDA<dtype>::cos(dtype* result, size_t num_elements) {
    applyUnaryOperation<cosFunc<dtype>>(result, num_elements);
}

template <typename dtype>
__device__ dtype expFunc(dtype x) {
    if constexpr (std::is_same<dtype, int>::value || std::is_same<dtype, int8_t>::value) {
        return static_cast<dtype>(exp(static_cast<float>(x)));
    } else if constexpr (std::is_same<dtype, __half>::value) {
        // If dtype is __half (half precision), use the CUDA-specific __hsin
        return __float2half(expf(__half2float(x)));
    } else {
        return exp(x);
    }
}

template <typename dtype>
void CUDA<dtype>::exp(dtype* result, size_t num_elements) {
    applyUnaryOperation<expFunc<dtype>>(result, num_elements);
}

template <typename dtype>
__device__ dtype logFunc(dtype x) {
    if constexpr (std::is_same<dtype, int>::value || std::is_same<dtype, int8_t>::value) {
        return static_cast<dtype>(log(static_cast<float>(x)));
    } else if constexpr (std::is_same<dtype, __half>::value) {
        // If dtype is __half (half precision), use the CUDA-specific __hsin
        return __float2half(logf(__half2float(x)));
    } else {
        return log(x);
    }
}

template <typename dtype>
void CUDA<dtype>::log(dtype* result, size_t num_elements) {
    applyUnaryOperation<logFunc<dtype>>(result, num_elements);
}

template <typename dtype>
__device__ dtype absFunc(dtype x) {
    if constexpr (std::is_same<dtype, __half>::value) {
        return __float2half(abs(__half2float(x)));
    } else {
        return abs(x);
    }
}

template <typename dtype>
void CUDA<dtype>::abs(dtype* result, size_t num_elements) {
    applyUnaryOperation<absFunc<dtype>>(result, num_elements);
}

template <typename dtype>
__device__ dtype tanhFunc(dtype x) {
    if constexpr (std::is_same<dtype, int>::value || std::is_same<dtype, int8_t>::value) {
        return static_cast<dtype>(tanh(static_cast<float>(x)));
    } else if constexpr (std::is_same<dtype, __half>::value) {
        // If dtype is __half (half precision), use the CUDA-specific __hsin
        return __float2half(tanhf(__half2float(x)));
    }  else {
        return tanh(x);
    }
}

template <typename dtype>
void CUDA<dtype>::tanh(dtype* result, size_t num_elements) {
    applyUnaryOperation<tanhFunc<dtype>>(result, num_elements);
}

template <typename dtype>
__device__ dtype siluFunc(dtype x) {
    // if constexpr (std::is_same<dtype, int>::value || std::is_same<dtype, int8_t>::value) {
    if constexpr (std::is_same<dtype, int>::value || std::is_same<dtype, int8_t>::value || std::is_same_v<dtype, half>) {
        return static_cast<dtype>(static_cast<float>(x) * (1 / (1 + exp(-static_cast<float>(x)))));
    } else {
        return x * (1 / (1 + exp(-x)));
    }
}

template <typename dtype>
void CUDA<dtype>::silu(dtype* result, size_t num_elements) {
    applyUnaryOperation<siluFunc<dtype>>(result, num_elements);
}

template <typename dtype>
__device__ dtype sqrtFunc(dtype x) {
    if constexpr (std::is_same<dtype, __half>::value) {
        // For __half, first convert to float for comparison
        if (__half2float(x) >= 0) {
            return __float2half(sqrtf(__half2float(x)));
        } else {
            return __float2half(nanf("")); // Return NaN for negative input
        }
    } else {
        if (x >= 0) {
            if constexpr (std::is_same<dtype, int>::value || std::is_same<dtype, int8_t>::value) {
                return static_cast<dtype>(sqrt(static_cast<float>(x)));
            } else {
                return sqrt(x); // Rsqrt calculation
            }
        } else {
            return nan("");
        }
    }
}

template <typename dtype>
void CUDA<dtype>::sqrt(dtype* result, size_t num_elements) {
    applyUnaryOperation<sqrtFunc<dtype>>(result, num_elements);
}

template <typename dtype>
__device__ dtype rsqrtFunc(dtype x) {
    if constexpr (std::is_same<dtype, __half>::value) {
        // For __half, first convert to float for comparison
        if (__half2float(x) > 0) {
            return __float2half(rsqrt(__half2float(x)));
        } else {
            return __float2half(nanf("")); // Return NaN for negative input
        }
    } else { 
        if (x > 0) {
            if constexpr (std::is_same<dtype, int>::value || std::is_same<dtype, int8_t>::value) {
                return static_cast<dtype>(rsqrt(static_cast<float>(x)));
            } else {
                return rsqrt(x); // Rsqrt calculation
            }
        } else {
            return nan("");
        }
    }
}

template <typename dtype>
void CUDA<dtype>::rsqrt(dtype* result, size_t num_elements) {
    applyUnaryOperation<rsqrtFunc<dtype>>(result, num_elements);
}

////////////////////////////////////////////////////// binary operations ///////////////////////////////////////////////////////////////////////////////
template <typename dtype> static inline __device__ dtype addFunc(dtype x, dtype y) { 
    if constexpr (std::is_same_v<dtype, __half>) {
        return __hadd(x, y); // Use CUDA's __haddfor addition}
    }
    return x + y; 
}
template <typename dtype> static inline __device__ dtype subFunc(dtype x, dtype y) { return x - y; }
template <typename dtype> static inline __device__ dtype mulFunc(dtype x, dtype y) { 
    if constexpr (std::is_same<dtype, __half>::value) {
        return __hmul(x, y); // Use CUDA's __hmul for multiplication
    }
    return x * y; 
}
template <typename dtype> static inline __device__ dtype divFunc(dtype a, dtype b) {
    // Handle comparison for __half type
    if constexpr (std::is_same<dtype, __half>::value) {
        if (__heq(b, __float2half(0.0f))) { // Use CUDA's __heq for comparison
            return __float2half(nanf("")); // Return NaN for division by zero
        }
        return __hdiv(a, b); // Use CUDA's __hdiv for division
    }
    // Handle comparison for other types
    else if (b == 0) {
        return nan(""); // Return NaN for division by zero
    }
    return a / b; // Default division for other types
}

// template <typename dtype> static inline __device__ dtype powFunc(dtype a, dtype b) { return pow(a, b); }
template <typename dtype> static inline __device__ dtype powFunc(dtype a, dtype b) { 
    if constexpr (std::is_same<dtype, __half>::value) {
        return __float2half(pow(__half2float(a), __half2float(b))); 
    } else {
        return pow(a, b); 
    }
}

template <typename dtype, dtype (*op)(dtype, dtype)>
__global__ void binaryKernel(dtype* result, const dtype* src1, const dtype* src2, size_t num_elements) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < num_elements) {
        result[i] = op(src1[i], src2[i]);
    }
}

template <typename dtype>
template <dtype (*op)(dtype, dtype)>
void CUDA<dtype>::applyBinaryOperation(dtype* result,  const dtype* other, size_t num_elements) const {
    int blockSize = 256;  // Number of threads per block (adjust based on optimization needs)
    int gridSize = (num_elements + blockSize - 1) / blockSize;  // Number of blocks
    binaryKernel<dtype, op><<<gridSize, blockSize>>>(result, this->data_, other, num_elements);
    CUDA_CHECK(hipGetLastError());
    // CUDA_CHECK(hipDeviceSynchronize());
}

template <typename dtype, dtype (*op)(dtype, dtype)>
__global__ void binaryScalarKernel(dtype* result, const dtype* src1, dtype value, size_t num_elements) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < num_elements) {
        result[i] = op(src1[i], value);
    }
}

template <typename dtype>
template <dtype (*op)(dtype, dtype)>
void CUDA<dtype>::applyBinaryScalarOperation(dtype* result,  dtype value, size_t num_elements) const {
    int blockSize = 256;  // Number of threads per block (adjust based on optimization needs)
    int gridSize = (num_elements + blockSize - 1) / blockSize;  // Number of blocks
    binaryScalarKernel<dtype, op><<<gridSize, blockSize>>>(result, this->data_, value, num_elements);
    CUDA_CHECK(hipGetLastError());
    // CUDA_CHECK(hipDeviceSynchronize());
}

template <typename dtype> void CUDA<dtype>::add(dtype* result, dtype* other, size_t num_elements) const {applyBinaryOperation<addFunc<dtype>>(result, other, num_elements);}
template <typename dtype> void CUDA<dtype>::sub(dtype* result, dtype* other, size_t num_elements) const {applyBinaryOperation<subFunc<dtype>>(result, other, num_elements);}
template <typename dtype> void CUDA<dtype>::mul(dtype* result, dtype* other, size_t num_elements) const {applyBinaryOperation<mulFunc<dtype>>(result, other, num_elements);}
template <typename dtype> void CUDA<dtype>::div(dtype* result, dtype* other, size_t num_elements) const {applyBinaryOperation<divFunc<dtype>>(result, other, num_elements);}

template <typename dtype> void CUDA<dtype>::add(dtype* result, dtype value, size_t num_elements) const {applyBinaryScalarOperation<addFunc<dtype>>(result, value, num_elements);}
template <typename dtype> void CUDA<dtype>::sub(dtype* result, dtype value, size_t num_elements) const {applyBinaryScalarOperation<subFunc<dtype>>(result, value, num_elements);}
template <typename dtype> void CUDA<dtype>::mul(dtype* result, dtype value, size_t num_elements) const {applyBinaryScalarOperation<mulFunc<dtype>>(result, value, num_elements);}
template <typename dtype> void CUDA<dtype>::div(dtype* result, dtype value, size_t num_elements) const {applyBinaryScalarOperation<divFunc<dtype>>(result, value, num_elements);}
template <typename dtype> void CUDA<dtype>::pow(dtype* result, dtype value, size_t num_elements) const {applyBinaryScalarOperation<powFunc<dtype>>(result, value, num_elements);}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

// template <typename dtype>
// __global__ void apply_rotary_emb_kernel(const dtype* input, dtype* result, int start_pos, int H, int W) {
//     int i = blockIdx.x * blockDim.x + threadIdx.x;  // Row index
//     int j = (blockIdx.y * blockDim.y + threadIdx.y) * 2;  // Column index (step by 2 for paired elements)
// 
//     if (i < H && j < W) {
//         int offset = i * W;
//         dtype theta = start_pos * 1.0f / pow(10000.0f, static_cast<dtype>(j) / static_cast<dtype>(W));
//         dtype cos_theta = cosf(theta); // only accept float for now
//         dtype sin_theta = sinf(theta);
// 
//         dtype v0 = input[offset + j];
//         dtype v1 = input[offset + j + 1];
// 
//         dtype rotary_emb_real = v0 * cos_theta - v1 * sin_theta;
//         dtype rotary_emb_imag = v0 * sin_theta + v1 * cos_theta;
// 
//         result[offset + j] = rotary_emb_real;
//         result[offset + j + 1] = rotary_emb_imag;
//     }
// }
// 
// template <typename dtype>
// void CUDA<dtype>::apply_rotary_emb(const dtype* input, dtype* result, int start_pos, int H, int W) const {
//     dim3 threadsPerBlock(16, 16);  // Define block size (16x16 is a typical choice, can be adjusted)
//     dim3 numBlocks((H + threadsPerBlock.x - 1) / threadsPerBlock.x,
//                    (W / 2 + threadsPerBlock.y - 1) / threadsPerBlock.y);  // Divide by 2 for W because j increments by 2
// 
//     apply_rotary_emb_kernel<<<numBlocks, threadsPerBlock>>>(input, result, start_pos, H, W);
//     CUDA_CHECK(hipGetLastError());
//     // CUDA_CHECK(hipDeviceSynchronize());
// }

template <typename dtype>
__global__ void apply_rotary_emb_kernel(const dtype* input, dtype* result, int start_pos, int B, int T, int n_heads, int head_dim) {
    int b = blockIdx.x;
    int t = blockIdx.y;
    int h = blockIdx.z;
    int d = threadIdx.x * 2;

    int offset = b * T * n_heads * head_dim + t * n_heads * head_dim + h * head_dim + d;

    float theta = (start_pos + t) * 1.0f / pow(10000.0f, static_cast<float>(d) / static_cast<float>(head_dim));
    // dtype theta = (start_pos + t) * 1.0f / pow(10000.0f, static_cast<dtype>(d) / static_cast<dtype>(head_dim));
    // dtype theta;
    // if constexpr (std::is_same_v<dtype, half>) {
    //     theta = (start_pos + t) * 1.0f / pow(10000.0f, __half2float(static_cast<float>(d)) / __half2float(static_cast<float>(head_dim)));
    // } else {
    //     theta = (start_pos + t) * 1.0f / pow(10000.0f, static_cast<dtype>(d) / static_cast<dtype>(head_dim));
    // }

    // dtype cos_theta = cosf(theta); // only accept float for now
    // dtype sin_theta = sinf(theta);

    dtype cos_theta = static_cast<dtype>(cos(theta));
    dtype sin_theta = static_cast<dtype>(sin(theta));

    dtype v0 = input[offset];
    dtype v1 = input[offset + 1];

    dtype rotary_emb_real = v0 * cos_theta - v1 * sin_theta;
    dtype rotary_emb_imag = v0 * sin_theta + v1 * cos_theta;

    result[offset] = rotary_emb_real;
    result[offset + 1] = rotary_emb_imag;
}

template <typename dtype>
void CUDA<dtype>::apply_rotary_emb(const dtype* input, dtype* result, int start_pos, int B, int T, int n_heads, int head_dim) const {
    dim3 grid(B, T, n_heads);
    dim3 block(head_dim / 2);

    apply_rotary_emb_kernel<<<grid, block>>>(input, result, start_pos, B, T, n_heads, head_dim);
    CUDA_CHECK(hipGetLastError());
    // CUDA_CHECK(hipDeviceSynchronize());
}

// CUDA kernel for type casting
template <typename dtype, typename OtherType>
__global__ void type_cast_kernel(dtype* result, const OtherType* src, size_t num_elements) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < num_elements) {
        if constexpr (std::is_same_v<dtype, half>) {
            result[i] = __float2half(static_cast<float>(src[i]));
        } else {
            result[i] = static_cast<dtype>(src[i]);
        }
    }
}

template <typename dtype>
template <typename OtherType>
void CUDA<dtype>::type_cast(dtype* result, const OtherType* src, size_t num_elements) {
    // Define block and grid sizes
    size_t blockSize = 256;
    size_t numBlocks = (num_elements + blockSize - 1) / blockSize;

    // Launch the CUDA kernel
    type_cast_kernel<<<numBlocks, blockSize>>>(result, src, num_elements);

    CUDA_CHECK(hipGetLastError());
    // CUDA_CHECK(hipDeviceSynchronize());
}
// Explicit instantiation of the template function for specific types
template void CUDA<float>::type_cast<float>(float*, const float*, size_t);
template void CUDA<float>::type_cast<int>(float*, const int*, size_t);
template void CUDA<float>::type_cast<half>(float*, const half*, size_t);

template void CUDA<int>::type_cast<float>(int*, const float*, size_t);
template void CUDA<int>::type_cast<int>(int*, const int*, size_t);
template void CUDA<int>::type_cast<half>(int*, const half*, size_t);

template void CUDA<half>::type_cast<float>(half*, const float*, size_t);
template void CUDA<half>::type_cast<int>(half*, const int*, size_t);
template void CUDA<half>::type_cast<half>(half*, const half*, size_t);

