#include "hip/hip_runtime.h"
#include "device/CUDA.hpp"
#include <cassert>
#include <cstddef>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/library_types.h>
#include <iostream>
#include <vector>
#include "util.hpp"

template class CUDA<float>;
// template class CUDA<int>;
// template class CUDA<int8_t>;
// Explicit template instantiation for different types
// template void CUDA<float>::rms_norm<float>(float* output, float* input, float* weight, float epsilon, int hidden_size, int num_tokens);
// template void CUDA<int>::rms_norm<int>(int* output, int* input, int* weight, float epsilon, int hidden_size, int num_tokens);
// template<> void CUDA<float>::rms_norm(float* output, float* input, float* weight, float epsilon, int hidden_size, int num_tokens);
// template<> void CUDA<int>::rms_norm(int* output, int* input, int* weight, float epsilon, int hidden_size, int num_tokens);

/*********************************************************************************************************************/
#define HandleNum 1 // the number of elements that each thread handles
/** the precision error is too big ( > 1e-3) compared to pytorch version */
template<typename dtype>
__global__ void rms_norm_kernel_v0(dtype *output, dtype *input, dtype *weight, float epsilon, int hidden_size) {
    const int bidx = blockIdx.x;
    const int tidx = threadIdx.x;

    // printf("bidx: %d, tidx: %d\n", bidx, tidx);

    // extern __shared__ dtype shared_mem[];
    // dtype* input_mem = shared_mem;
    // dtype* input2_mem = shared_mem + hidden_size; // used for parallel reduction
    // dtype* weight_mem = shared_mem + 2 * hidden_size;
    // __shared__ dtype rms;

    extern __shared__ dtype shared_mem[];
    // dtype input_mem[hidden_size];
    dtype* input2_mem = shared_mem; // used for parallel reduction
    // dtype weight_mem[hidden_size];
    __shared__ dtype rms;

    // fetch input and weight into shared memory
    // maybe have bank conflict here!!! because 1 thread access continuous address in shared memory
    for (int i = tidx * HandleNum; i < (tidx + 1) * HandleNum; i++) {
        if (i < hidden_size) {
            // input_mem[i]  = input[bidx * hidden_size + i];
            dtype val = input[bidx * hidden_size + i];
            // printf("input[%d]: %f\n", i, input_mem[i]);
            // input2_mem[i] = input_mem[i] * input_mem[i];
            input2_mem[i] = val * val;
            // weight_mem[i] = weight[i];
        }
    }

    // parallel reduction to calculate the sum of input * input
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        __syncthreads();
        if (tidx < stride) {
            for (int i = 0; i < HandleNum; i++) {
                input2_mem[tidx * HandleNum + i] += input2_mem[(tidx + stride) * HandleNum + i];
            }
        }   
    }

    // calculate rms
    if (tidx == 0) {
        rms = 0.0; // forget to initialize rms cause the precision error!!!!!!!!!!!!!!!!!!!
        for (int i = 0; i < HandleNum; i++) {
            rms += input2_mem[i];
        }
        rms = sqrtf(rms / hidden_size + epsilon);
        // rms = sqrt(rms / hidden_size + epsilon);
        // dtype rms_r = rsqrtf()
    }

    __syncthreads();

    // calculate output
    for (int i = tidx * HandleNum; i < (tidx + 1) * HandleNum; i++) {
        if (i < hidden_size) {
            // dtype val = input_mem[i];
            dtype val = input[bidx * hidden_size + i];
            dtype w = weight[i];
            output[bidx * hidden_size + i] = val * w / rms;
            // printf("bidx: %d, tid: %d, val: %f, w: %f, rms: %f\n", bidx, tidx, val, w, rms);
            // printf("output[%d]: %f\n", i, output[bidx * hidden_size + i]);
        }
    }
}

/**
 * a usual shape is (batch_size, sequence_length, hidden_size)
 * fused rms norm kernel
 * @tparam dtype 
 * num_tokens: number of tokens, batch size * sequence length
 */
template<typename dtype>
void rms_norm_v0(dtype *output, dtype *input, dtype *weight, float epsilon, int hidden_size, int num_tokens) {
    assert(hidden_size % HandleNum == 0);
    int gridSize = num_tokens;
    int blockSize = hidden_size / HandleNum;

    // assert(blockSize <= 1024);
    // assert(gridSize <= 1024);

    // get the max block size and grid size
    int blockSizeLimit;
    hipDeviceGetAttribute(&blockSizeLimit, hipDeviceAttributeMaxThreadsPerBlock, 0);
    // std::cout << "Block Size Limit: " << blockSizeLimit << std::endl;
    assert(blockSize <= blockSizeLimit);

    int gridSizeLimit;
    hipDeviceGetAttribute(&gridSizeLimit, hipDeviceAttributeMaxGridDimX, 0);
    // std::cout << "Grid Size Limit: " << gridSizeLimit << std::endl;
    assert(gridSize <= gridSizeLimit);


    int shared_mem_size = hidden_size * sizeof(dtype);
    int smem;
    hipDeviceGetAttribute(&smem, hipDeviceAttributeMaxSharedMemoryPerBlock, 0);
    // std::cout << "Shared Memory Limit: " << smem << " bytes" << std::endl;
    assert(shared_mem_size + sizeof(dtype) <= smem); // 1 more dtype for rms
    
    rms_norm_kernel_v0<<<gridSize, blockSize, shared_mem_size>>>(output, input, weight, epsilon, hidden_size);

    CUDA_CHECK(hipGetLastError()); // if shared memory is not enough or grid / block size too large, it will return an error here.
    CUDA_CHECK(hipDeviceSynchronize());
}

/*********************************************************************************************************************/
// reference: https://github.com/karpathy/llm.c
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
namespace cg = cooperative_groups;

// version 1 of rms norm kernel, one warp handles one token row
template <typename dtype>
__global__ void rms_norm_kernel_v1(dtype *output, dtype *input, dtype *weight, float epsilon, int hidden_size, int num_tokens) {
    cg::thread_block block = cg::this_thread_block();
    cg::thread_block_tile<32> warp = cg::tiled_partition<32>(block);
    int idx = blockIdx.x * warp.meta_group_size() + warp.meta_group_rank(); // the warp index, the row(token) index
    if (idx >= num_tokens) 
        return;

    // the row of input that this group of threads will process
    const dtype *x = input + idx * hidden_size;

    // mean
    dtype sum = 0.0f; // aussume dtype is float
    for (int i = warp.thread_rank(); i < hidden_size; i += warp.size()) {
        sum += x[i] * x[i];
    }

    sum = cg::reduce(warp, sum, cg::plus<dtype>()); // sum of all elements in the row

    // dtype mean = sum / hidden_size;

    // sum = 0.0f;
    dtype rms = sqrtf(sum / hidden_size + epsilon);

    dtype *o = output + idx * hidden_size;
    for (int i = warp.thread_rank(); i < hidden_size; i += warp.size()) {
        o[i] = x[i] * weight[i] / rms;
    }
}

template <typename dtype>
void rms_norm_v1(dtype *output, dtype *input, dtype *weight, float epsilon, int hidden_size, int num_tokens) {
    int blockSize = 256;
    int gridSize = div_ceil(num_tokens * 32, blockSize);
    rms_norm_kernel_v1<<<gridSize, blockSize>>>(output, input, weight, epsilon, hidden_size, num_tokens);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
}

/********************************************************* rms_norm ****************************************************/

template<typename dtype>
void CUDA<dtype>::rms_norm(dtype *output, dtype *input, dtype *weight, float epsilon, int hidden_size, int num_tokens) {
    rms_norm_v0(output, input, weight, epsilon, hidden_size, num_tokens);
    // rms_norm_v1(output, input, weight, epsilon, hidden_size, num_tokens);
}
