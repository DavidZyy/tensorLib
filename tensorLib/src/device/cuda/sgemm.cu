// this file provides single precision matrix multiplication kernels(Sgemm).
#include "device/cuda/CUDA.cuh"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <iostream>

// may have error...
void sgemm_cublas(const float* lhs, const float* rhs, float* result, size_t M, size_t N, size_t K) {
        hipblasHandle_t handle;
        CUBLAS_CHECK(hipblasCreate(&handle));

        float alpha = 1.0f;
        float beta = 0.0f;

        CUBLAS_CHECK(hipblasSgemm(
            handle, 
            HIPBLAS_OP_T, HIPBLAS_OP_N, 
            N, M, K, 
            &alpha, 
            rhs, N, 
            lhs, K, 
            &beta, 
            result, N));

        CUBLAS_CHECK(hipblasDestroy(handle));
}
