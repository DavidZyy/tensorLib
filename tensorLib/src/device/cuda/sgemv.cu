#include "hip/hip_runtime.h"
// reference: git@github.com:Bruce-Lee-LY/cuda_hgemv.git
// in decode stage of transformer, the operation is gemv, which is a matrix-vector multiplication
#include "device/CUDA.hpp"
#include <cassert>
#include <cstddef>
#include <cstdint>
#include <cstdio>
#include <iostream>
#include "util.hpp"

template class CUDA<float>;
template class CUDA<int>;
template class CUDA<int8_t>;

/************************************************************************************************************************************************************/
size_t initThreadSmem(size_t K) {
    int dev_id = 0;
    CUDA_CHECK(hipGetDevice(&dev_id));

    hipDeviceProp_t dev_prop;
    CUDA_CHECK(hipGetDeviceProperties(&dev_prop, dev_id));

    size_t smem_max_size = K * sizeof(float);
    // assert(dev_prop.sharedMemPerMultiprocessor >= smem_max_size);
    assert(dev_prop.sharedMemPerBlock >= smem_max_size);
    // printf("Shared memory per multiprocessor: %zu\n", dev_prop.sharedMemPerMultiprocessor);
    // printf("Max shared memory per block: %zu\n", dev_prop.sharedMemPerBlock);
    // printf("smem_max_size: %zu\n", smem_max_size);

    // CUDA_CHECK(hipFuncSetAttribute(reinterpret_cast<const void*>(gemvKernelV1<float>), hipFuncAttributeMaxDynamicSharedMemorySize, smem_max_size));

    return smem_max_size;
}

/************************************************************************************************************************************************************/
// // threads
// #define WARP_SIZE 32
// #define WARPS_PER_BLOCK 4
// #define THREADS_PER_BLOCK 128  // WARP_SIZE * WARPS_PER_BLOCK
// template<typename dtype>
// __global__ void gemvKernelV0(const dtype* A, const dtype* B, dtype* C, size_t M, size_t N, size_t K) {
//     const size_t col = blockIdx.x * THREADS_PER_BLOCK + threadIdx.x;
//     if (col >= N) return;
// 
//     float tmp = 0.0;
// 
//     #pragma unroll
//     for (size_t i = 0; i < K; ++i) {
//         tmp += A[i] * B[i + col * K]; // B is col major
//         // tmp += A[i] * B[i*N + col];  // B is row major
//     }
// 
//     C[col] = tmp;
// }
// 
// template<typename dtype>
// void gemvV0(const dtype* A, const dtype* B, dtype* C, size_t M, size_t N, size_t K) {
//     dim3 block(THREADS_PER_BLOCK);
//     dim3 grid(div_ceil(N, THREADS_PER_BLOCK));
//     gemvKernelV0<<<grid, block>>>(A, B, C, M, N, K);
//     CUDA_CHECK(hipGetLastError());
//     CUDA_CHECK(hipDeviceSynchronize());
// }
// 
// template void gemvV0<float>(const float* A, const float* B, float* C, size_t M, size_t N, size_t K);
// template void gemvV0<int>(const int* A, const int* B, int* C, size_t M, size_t N, size_t K);
// template void gemvV0<int8_t>(const int8_t* A, const int8_t* B, int8_t* C, size_t M, size_t N, size_t K);

/************************************************************************************************************************************************************/
// threads + shared memory
#define WARP_SIZE 32
#define WARPS_PER_BLOCK 4
#define THREADS_PER_BLOCK 128  // WARP_SIZE * WARPS_PER_BLOCK
template<typename dtype>
__global__ void gemvKernelV1(const dtype* A, const dtype* B, dtype* C, size_t M, size_t N, size_t K) {
    // printf("k: %d\n", K);
    extern __shared__ float A_smem[];
    // __shared__ float A_smem[2048];
    // assert(K <= 2048);

    size_t A_smem_iters = div_ceil(K, THREADS_PER_BLOCK);

    // fetch A from global memory to shared memory
    # pragma unroll
    for (size_t i = 0; i < A_smem_iters; ++i) {
        size_t idx = i * THREADS_PER_BLOCK + threadIdx.x;
        if (idx < K) {
            A_smem[idx] = A[idx];
        }
    }

    __syncthreads();

    const size_t col = blockIdx.x * THREADS_PER_BLOCK + threadIdx.x;
    if (col >= N) return;

    float tmp = 0.0;
    # pragma unroll
    for (size_t i = 0; i < K; ++i) {
        tmp += A[i] * B[i + col * K]; // B is col major
        // tmp += A_smem[i] * B[i + col * K]; // B is col major
        // tmp += A_smem[i] * B[i*N + col];  // B is row major
        // tmp += B[i*N + col];  // B is row major
    }

    C[col] = tmp;
}

template <typename dtype>
void gemvV1(const dtype* A, const dtype* B, dtype* C, size_t M, size_t N, size_t K) {
    static size_t smem_size = initThreadSmem(K);

    dim3 block(THREADS_PER_BLOCK);
    dim3 grid(div_ceil(N, THREADS_PER_BLOCK));
    // gemvKernelV1<<<grid, block>>>(A, B, C, M, N, K);
    gemvKernelV1<<<grid, block, K * sizeof(dtype)>>>(A, B, C, M, N, K);
    // gemvKernelV1<<<grid, block, 1024*2>>>(A, B, C, M, N, K);
    // gemvKernelV1<<<grid, block, smem_size>>>(A, B, C, M, N, K);
    // static size_t smm = 4*256;
    // gemvKernelV1<<<grid, block, smm>>>(A, B, C, M, N, K);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
}

template void gemvV1<float>(const float* A, const float* B, float* C, size_t M, size_t N, size_t K);
template void gemvV1<int>(const int* A, const int* B, int* C, size_t M, size_t N, size_t K);
template void gemvV1<int8_t>(const int8_t* A, const int8_t* B, int8_t* C, size_t M, size_t N, size_t K);

/************************************************************************************************************************************************************/
#define WARP_SIZE 32
#define WARPS_PER_BLOCK 4
#define THREADS_PER_BLOCK 128  // WARP_SIZE * WARPS_PER_BLOCK
template <typename dtype>
__global__ void gemv_kernel_v2(const dtype* A, const dtype* B, dtype* C, size_t N, size_t K) {
    const size_t warp_id = threadIdx.x / WARP_SIZE; // the warp index which this thread belongs to
    const size_t warp_col = blockIdx.x * WARPS_PER_BLOCK + warp_id; // the column index this warp will process

    if (warp_col >= N) return;

    const size_t K_iters = div_ceil(K, WARP_SIZE); // the number of iterations to process all elements in A
    const size_t lane_id = threadIdx.x % WARP_SIZE; // the lane index in the warp

    dtype tmp = 0.0;
#pragma unroll
    for (size_t i = 0; i < K_iters; ++i) {
        size_t A_idx = i * WARP_SIZE + lane_id;
        size_t B_idx = i * WARP_SIZE + lane_id + warp_col * K;
        if (A_idx < K) {
            tmp += A[A_idx] * B[B_idx];
        }
    }

    constexpr unsigned int mask = 0xffffffff;
#pragma unroll
    for (size_t i = WARP_SIZE / 2; i >= 1; i /= 2) {
        tmp += __shfl_xor_sync(mask, tmp, i);
    }

    if (lane_id == 0) {
        C[warp_col] = tmp;
    }
}

template<typename dtype>
void gemv_v2(const dtype* A, const dtype* B, dtype* C, size_t N, size_t K) {
    dim3 block(THREADS_PER_BLOCK);
    dim3 grid(div_ceil(N, WARPS_PER_BLOCK));
    gemv_kernel_v2<<<grid, block>>>(A, B, C, N, K);
}

template void gemv_v2<float>(const float* A, const float* B, float* C, size_t N, size_t K);
template void gemv_v2<int>(const int* A, const int* B, int* C, size_t N, size_t K);
template void gemv_v2<int8_t>(const int8_t* A, const int8_t* B, int8_t* C, size_t N, size_t K);

/************************************************************************************************************************************************************/
